#include "hip/hip_runtime.h"
#include <vector>

#include "utils.h"

// trt_reduce
using fptr_t = int64_t;
fptr_t init_custom_ar(int64_t rank_id, int64_t world_size, torch::Tensor& rank_data, const std::vector<fptr_t>& buffers,
                      const std::vector<fptr_t>& tmp_result_buffers, const std::vector<fptr_t>& barrier_in,
                      const std::vector<fptr_t>& barrier_out);
void dispose(fptr_t _fa);
void all_reduce(fptr_t _fa, torch::Tensor& inp, torch::Tensor& out);
std::tuple<std::vector<int64_t>, std::vector<int64_t>> get_graph_buffer_ipc_meta(fptr_t _fa);
void register_graph_buffers(fptr_t _fa, const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets);

// moe_align_block_size
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad,
                          torch::Tensor token_cnts_buffer, torch::Tensor cumsum_buffer);

// sampling_scaling_penalties
torch::Tensor sampling_scaling_penalties(const torch::Tensor& logits, const torch::Tensor& scaling_penalties);

// int8_scaled_mm
torch::Tensor int8_scaled_mm(const torch::Tensor& mat_a, const torch::Tensor& mat_b, const torch::Tensor& scales_a,
                             const torch::Tensor& scales_b, const torch::Dtype& out_dtype,
                             const c10::optional<torch::Tensor>& bias);

// rotary embedding
void rotary_embedding(torch::Tensor& positions, torch::Tensor& query, torch::Tensor& key, int64_t head_size,
                      torch::Tensor& cos_sin_cache, bool is_neox);

// rms norm
void rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps, int64_t hip_stream);

// fused rms norm
void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps, int64_t hip_stream);

// gemma rms norm
void gemma_rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps, int64_t hip_stream);

// fused gemma rms norm
void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                             int64_t hip_stream);

// silu and mul
void silu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// gelu tanh and mul
void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// gelu and mul
void gelu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// bmm fp8
void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // trt_reduce
  m.def("init_custom_ar", &init_custom_ar, "init custom allreduce meta (CUDA)");
  m.def("dispose", &dispose, "dispose custom allreduce meta");
  m.def("all_reduce", &all_reduce, "custom all reduce (CUDA)");
  m.def("get_graph_buffer_ipc_meta", &get_graph_buffer_ipc_meta, "custom all reduce get graph ipc meta");
  m.def("register_graph_buffers", &register_graph_buffers, "custom all reduce register graph buffers");
  // moe_align_block_size
  m.def("moe_align_block_size", &moe_align_block_size, "MOE Align Block Size (CUDA)");
  // sampling_scaling_penalties
  m.def("sampling_scaling_penalties", &sampling_scaling_penalties, "Sampling scaling penalties (CUDA)");
  // int8_scaled_mm
  m.def("int8_scaled_mm", &int8_scaled_mm, "INT8 scaled matmul (CUDA)");
  // rotary embedding
  m.def("rotary_embedding", &rotary_embedding, "Rotary Embedding (CUDA)");
  // rms norm
  m.def("rmsnorm", &rmsnorm, "RMSNorm (CUDA)");
  // fused rms norm
  m.def("fused_add_rmsnorm", &fused_add_rmsnorm, "Fused Add RMSNorm (CUDA)");
  // gemma rms norm
  m.def("gemma_rmsnorm", &gemma_rmsnorm, "Gemma RMSNorm (CUDA)");
  // fused gemma rms norm
  m.def("gemma_fused_add_rmsnorm", &gemma_fused_add_rmsnorm, "Gemma Fused Add RMSNorm (CUDA)");
  // silu and mul
  m.def("silu_and_mul", &silu_and_mul, "Silu and Mul (CUDA)");
  // gelu tanh and mul
  m.def("gelu_tanh_and_mul", &gelu_tanh_and_mul, "Gelu Tanh and Mul (CUDA)");
  // gelu and mul
  m.def("gelu_and_mul", &gelu_and_mul, "Gelu and Mul (CUDA)");
  // bmm fp8
  m.def("bmm_fp8", &bmm_fp8, "BMM FP8 (CUDA)");
}
